#include "hip/hip_runtime.h"
// CUDA kernel for Smith-Waterman sequence alignment
// Optimized for RTX 4070 (Ada Lovelace architecture, SM 8.9)

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Constants for RTX 4070 optimization
#define THREADS_PER_BLOCK 256
#define WARP_SIZE 32
#define MAX_BLOCKS 65535

// Shared memory for block-level reduction
__shared__ int shared_scores[THREADS_PER_BLOCK];

// Main Smith-Waterman alignment kernel
extern "C" __global__ void smith_waterman_kernel(
    const unsigned char* __restrict__ seq1,
    const unsigned char* __restrict__ seq2,
    int* __restrict__ result,
    const int sequence_length
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_score = 0;
    
    // Process multiple elements per thread for better occupancy
    const int elements_per_thread = 4;
    const int total_threads = gridDim.x * blockDim.x;
    
    for (int i = idx; i < sequence_length; i += total_threads) {
        // Load sequences with coalesced memory access
        unsigned char a = seq1[i];
        unsigned char b = seq2[i];
        
        // Smith-Waterman scoring: +2 for match, -1 for mismatch
        local_score += (a == b) ? 2 : -1;
    }
    
    // Store local score in shared memory
    shared_scores[threadIdx.x] = local_score;
    __syncthreads();
    
    // Block-level reduction using warp shuffle
    cg::thread_block block = cg::this_thread_block();
    
    // Reduce within warps first
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        local_score += __shfl_down_sync(0xffffffff, local_score, offset);
    }
    
    // Write warp results to shared memory
    if (threadIdx.x % WARP_SIZE == 0) {
        shared_scores[threadIdx.x / WARP_SIZE] = local_score;
    }
    __syncthreads();
    
    // Final reduction across warps (only first thread in block)
    if (threadIdx.x == 0) {
        int block_score = 0;
        int num_warps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
        
        for (int i = 0; i < num_warps; i++) {
            block_score += shared_scores[i];
        }
        
        // Atomic add to global result
        atomicAdd(result, block_score);
    }
}

// Optimized kernel for large sequences with memory coalescing
extern "C" __global__ void smith_waterman_large_kernel(
    const unsigned char* __restrict__ seq1,
    const unsigned char* __restrict__ seq2,
    int* __restrict__ result,
    const int sequence_length,
    const int chunk_size
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_score = 0;
    
    // Process chunks for better memory locality
    for (int chunk_start = 0; chunk_start < sequence_length; chunk_start += chunk_size) {
        int chunk_end = min(chunk_start + chunk_size, sequence_length);
        
        for (int i = chunk_start + tid; i < chunk_end; i += gridDim.x * blockDim.x) {
            unsigned char a = seq1[i];
            unsigned char b = seq2[i];
            local_score += (a == b) ? 2 : -1;
        }
    }
    
    // Block-level reduction
    shared_scores[threadIdx.x] = local_score;
    __syncthreads();
    
    // Warp-level reduction
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        local_score += __shfl_down_sync(0xffffffff, local_score, offset);
    }
    
    if (threadIdx.x % WARP_SIZE == 0) {
        shared_scores[threadIdx.x / WARP_SIZE] = local_score;
    }
    __syncthreads();
    
    if (threadIdx.x == 0) {
        int block_score = 0;
        int num_warps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
        
        for (int i = 0; i < num_warps; i++) {
            block_score += shared_scores[i];
        }
        
        atomicAdd(result, block_score);
    }
}

// Kernel for 16-file batch processing
extern "C" __global__ void smith_waterman_batch_kernel(
    const unsigned char* __restrict__ sequences,
    int* __restrict__ results,
    const int* __restrict__ sequence_lengths,
    const int* __restrict__ sequence_offsets,
    const int num_pairs
) {
    int pair_idx = blockIdx.x;
    if (pair_idx >= num_pairs) return;
    
    int seq1_offset = sequence_offsets[pair_idx * 2];
    int seq2_offset = sequence_offsets[pair_idx * 2 + 1];
    int seq1_len = sequence_lengths[pair_idx * 2];
    int seq2_len = sequence_lengths[pair_idx * 2 + 1];
    int min_len = min(seq1_len, seq2_len);
    
    int tid = threadIdx.x;
    int local_score = 0;
    
    // Process sequence pair
    for (int i = tid; i < min_len; i += blockDim.x) {
        unsigned char a = sequences[seq1_offset + i];
        unsigned char b = sequences[seq2_offset + i];
        local_score += (a == b) ? 2 : -1;
    }
    
    // Block reduction
    shared_scores[tid] = local_score;
    __syncthreads();
    
    // Warp reduction
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        local_score += __shfl_down_sync(0xffffffff, local_score, offset);
    }
    
    if (tid % WARP_SIZE == 0) {
        shared_scores[tid / WARP_SIZE] = local_score;
    }
    __syncthreads();
    
    if (tid == 0) {
        int block_score = 0;
        int num_warps = (blockDim.x + WARP_SIZE - 1) / WARP_SIZE;
        
        for (int i = 0; i < num_warps; i++) {
            block_score += shared_scores[i];
        }
        
        results[pair_idx] = block_score;
    }
}

// Memory management helpers
extern "C" __device__ void* allocate_shared_memory(size_t size) {
    extern __shared__ char shared_memory[];
    return shared_memory;
}

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            return; \
        } \
    } while(0) 